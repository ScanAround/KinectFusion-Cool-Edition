#include "hip/hip_runtime.h"
#include "../../cpu/frame/Frame.h"
#include "../../cpu/tsdf/voxel.h"
#include "../../cpu/tsdf/kinect_fusion_utility.h"
#include "../../cpu/mesher/Marching_Cubes.h"
#include <eigen3/Eigen/Dense>
#define MINF -std::numeric_limits<float>::infinity()

__global__ 
void initialize(kinect_fusion::Voxel *cu_grid, int dimX, int dimY, int dimZ, int dimYZ, Eigen::Vector3d voxelSize, dim3 thread_nums, Eigen::Vector3d center){
  int id_x = threadIdx.x + blockIdx.x * thread_nums.x;
  int id_y = threadIdx.y + blockIdx.y * thread_nums.y;
  int id_z = threadIdx.z + blockIdx.z * thread_nums.z;
  if(id_x < dimX && id_y < dimY && id_z < dimZ){
    cu_grid[id_x*dimYZ + id_y*dimZ + id_z].position = voxelSize.cwiseProduct(Eigen::Vector3d(id_x, id_y, id_z)) + voxelSize * 0.5 + center;
    cu_grid[id_x*dimYZ + id_y*dimZ + id_z].weight = 1.0;
    cu_grid[id_x*dimYZ + id_y*dimZ + id_z].tsdfValue = nan("1");
  }
}
__device__
double TSDF(double eta, double mu){
  if (eta >= -mu) return min(1.0, -eta / mu);
  return nan("1");
  // return 5000.0f;
}

__device__ 
Eigen::Vector2i vec_to_pixel(const Eigen::Vector3d vec, Eigen::Matrix3d R_i, Eigen::Vector3d t_i, Eigen::Matrix3d K_calibration, int width, int height){

  Eigen::Vector3d vec_camera_frame = R_i * vec + t_i;
  
  Eigen::Vector3d u_dot = (K_calibration * vec_camera_frame) / vec_camera_frame[2];

  Eigen::Vector2i u;
  if(u_dot[0] >= 0 
  && u_dot[0] <= width 
  && u_dot[1] >= 0 
  && u_dot[1] <= height){
      u << int(u_dot[0]), int(u_dot[1]);
  }
  else{
      u << -1, -1 ;
  }
  return u;
}
__device__
double projectiveTSDF(Eigen::Matrix3d K, Eigen::Matrix3d K_i,  Eigen::Vector3d p, Eigen::Matrix3d R_i, Eigen::Vector3d t_i, Eigen::Vector3d t, float *R, int width, int height, double mu){
  Eigen::Vector2i x = vec_to_pixel(p, R_i, t_i, K, width, height);

  // Compute lambda
  if(x[0] == -1 || x[1] == -1 || isnan(R[x[1]*width + x[0]]) || R[x[1]*width + x[0]] <= 0.0f) {
    return nan("1");
  }
  double lambda = (K_i * x.cast<double>().homogeneous()).norm();

  // Compute eta
  // we have to convert R_k values to meters
  double eta = (1.0 / lambda) * (t - p).norm() - static_cast<double>((R[x[1]*width + x[0]]) *255.0f* 255.0f / 5000.0f);

  // Compute TSDF value
  double F_R_k_p = TSDF(eta, mu);
  // Here, we return the TSDF value and the corresponding image coordinate.
  return F_R_k_p;
}

__global__
void update(kinect_fusion::Voxel *cu_grid,
            int dimX, int dimY, int dimZ, int dimYZ,
            Eigen::Vector3d voxelSize, Eigen::Matrix3d K, Eigen::Matrix3d K_i, Eigen::Matrix3d R_i, Eigen::Vector3d t_i, Eigen::Vector3d t, 
            float *R, int width, int height, double mu, 
            dim3 thread_nums){
  int id_x = threadIdx.x + blockIdx.x * thread_nums.x;
  int id_y = threadIdx.y + blockIdx.y * thread_nums.y;
  int id_z = threadIdx.z + blockIdx.z * thread_nums.z;
  if(id_x < dimX && id_y < dimY && id_z < dimZ){
    kinect_fusion::Voxel& voxel = cu_grid[id_x*dimYZ + id_y*dimZ + id_z];
    Eigen::Vector3d p(voxel.position); // The point in the global frame
    double F_R = projectiveTSDF(K, K_i, p, R_i, t_i, t, R, width, height, mu);
    if(!isnan(F_R)){
      if(isnan(voxel.tsdfValue)){
        voxel.tsdfValue = F_R;
      }
      else{
        voxel.tsdfValue = (voxel.tsdfValue * voxel.weight + F_R) / (voxel.weight + 1.0);
      }
      voxel.weight += 1.0;
    }
  }
}

namespace kinect_fusion {

VoxelGrid::VoxelGrid(size_t dimX, size_t dimY, size_t dimZ, Eigen::Vector3d gridSize_, Eigen::Vector3d ctr_of_mass) : 
                    dimX(dimX), dimY(dimY), dimZ(dimZ), dimYZ(dimY*dimZ), gridSize(gridSize_), 
                    center(-0.5 * gridSize_ + ctr_of_mass) {
                    // center(-0.5*gridSize_){
  center[2] = 0.0f;
  grid.resize(dimX * dimYZ);
  voxelSize = gridSize.cwiseQuotient(Eigen::Vector3d(dimX, dimY, dimZ));
  initializeGrid();
}

void VoxelGrid::initializeGrid() {
  hipError_t cudaStatus = hipMallocManaged(&cu_grid, dimX * dimYZ * sizeof(Voxel));
  if(cudaStatus != hipSuccess){
    std::cout << "Problem in CudaMalloc: " << hipGetErrorString(cudaStatus) << std::endl;
  }
  const int tile_dim = 4; // make sure it's a multiple of dimX -> not sure what the optimal tile_dim is 
  dim3 thread_nums(tile_dim, tile_dim, tile_dim);  
  dim3 block_nums(dimX/tile_dim, dimY/tile_dim, dimZ/tile_dim);

  initialize <<<block_nums,thread_nums>>> (cu_grid, dimX, dimY, dimZ, dimYZ, voxelSize, thread_nums, center);
  hipDeviceSynchronize();

  hipError_t cudaStatus2 = hipMemcpy(grid.data(), cu_grid, dimX * dimYZ * sizeof(Voxel), hipMemcpyDeviceToHost);
  if(cudaStatus2 != hipSuccess){
    std::cout << "Problem in Copying: " << hipGetErrorString(cudaStatus2) << std::endl;
  };

  ddx = 1.0f / (dimX - 1);
  ddy = 1.0f / (dimY - 1);
  ddz = 1.0f / (dimZ - 1);

  max = grid[(dimX-1)*dimYZ + (dimY-1)*dimZ + (dimZ-1)].position;
  min = grid[0].position;
}

/*
Voxel& VoxelGrid::getVoxel(size_t x, size_t y, size_t z) {
  return grid[x*dimYZ + y*dimZ + z];
}

size_t VoxelGrid::getDimX() const {
  return dimX;
}

size_t VoxelGrid::getDimY() const {
  return dimY;
}

size_t VoxelGrid::getDimZ() const {
  return dimZ;
}*/

void VoxelGrid::updateGlobalTSDF(Frame& curr_frame,
                                 double mu) {

  float *R;
  auto CudaAssignemnt = hipMalloc(&R, sizeof(float) * curr_frame.width * curr_frame.height);
  if(CudaAssignemnt != hipSuccess){
    std::cout << "Problem in Assignment: " << CudaAssignemnt <<std::endl;
  }
  auto CudaCopy = hipMemcpy(R, curr_frame.Depth_k, sizeof(float) * curr_frame.width * curr_frame.height, hipMemcpyHostToDevice);
  if(CudaCopy != hipSuccess){
    std::cout << "Problem in Copying: " << CudaCopy <<std::endl;
  }

  const int tile_dim = 4; // make sure it's a multiple of dimX -> not sure what the optimal tile_dim is 
  dim3 thread_nums(tile_dim, tile_dim, tile_dim);  // maybe make this a class attribute
  dim3 block_nums(dimX/tile_dim, dimY/tile_dim, dimZ/tile_dim);
  
  auto K = curr_frame.K_calibration.cast<double>();
  auto K_i = K.inverse();
  auto T_gk = curr_frame.T_gk.cast<double>();
  auto R_i = T_gk.inverse().block(0,0,3,3);
  auto t_i = T_gk.inverse().block(0,3,3,1);
  auto t = T_gk.block(0,3,3,1);
  update <<<block_nums,thread_nums>>> (cu_grid, dimX, dimY, dimZ, dimYZ, 
                                       voxelSize, K , K_i, R_i, t_i, t,
                                       R, curr_frame.width, curr_frame.height, mu, thread_nums);
 hipDeviceSynchronize();
  
  hipError_t cudaStatus2 = hipMemcpy(grid.data(), cu_grid, dimX * dimYZ * sizeof(Voxel), hipMemcpyDeviceToHost);
  if(cudaStatus2 != hipSuccess){
    std::cout << "Problem in Copying: " << hipGetErrorString(cudaStatus2) << std::endl;
  };
  hipFree(R);
}

}


// int main(){

// auto pose_f = Eigen::Matrix4f::Identity();

// const char* img_loc = "/home/amroabuzer/Desktop/KinectFusion/KinectFusion-Cool-Edition/data/rgbd_dataset_freiburg1_xyz/depth/1305031102.160407.png"; 

// Frame* frame1 = new Frame(img_loc, pose_f, 1.0);

// frame1 -> process_image();

// frame1 -> save_G_off_format("G.obj");
// frame1 -> save_off_format("original.obj");

// Eigen::Vector3d gridSize(4,4,4); 
// unsigned int res = 256;
// Eigen::Vector3d ctr_of_mass = (frame1 -> center_of_mass).cast<double>();
// kinect_fusion::VoxelGrid grid(res ,res ,res ,gridSize, ctr_of_mass);
// double mu = 0.02;
// auto start = std::chrono::high_resolution_clock::now();
// grid.updateGlobalTSDF(*frame1, mu);

// auto end = std::chrono::high_resolution_clock::now();
// auto duration = std::chrono::duration_cast<std::chrono::seconds>(end - start).count();

// // kinect_fusion::utility::writeTSDFToFile("TSDF.txt", grid);
// Marching_Cubes::Mesher(grid, 0, "mesh.off");

// std::cout << "time for execution: " << duration << std::endl; 
// }