#include "hip/hip_runtime.h"
#include "../../cpu/frame/Frame.h"
#include "../../cpu/frame/Frame_Pyramid.h"
#include <iostream>
#include <fstream>
#include <FreeImage.h>
#include <vector>
#include <Eigen/Dense>
#define MINF -std::numeric_limits<float>::infinity()

#define MAXTHRESHOLD 10

__global__
void apply_bilateral_cuda(float* depthMap, float* filteredImage, int diameter, double sigmaS, double sigmaR, int width, int height) {
	double filtered = 0;
	double wP = 0;
	int neighbor_x = 0;
	int neighbor_y = 0;
	int half = diameter / 2;
	int id_x = blockIdx.x * blockDim.x + threadIdx.x;
	int id_y = blockIdx.y * blockDim.y + threadIdx.y;
	for (int i = 0; i < diameter; i++) {
		for (int j = 0; j < diameter; j++) {

			neighbor_x = id_x - (half - i);
			neighbor_y = id_y - (half - j);
			if (neighbor_x > 0 && neighbor_y > 0 && neighbor_x < height && neighbor_y < width)
			{
				if (depthMap[neighbor_x * width + neighbor_y] <= 0.0f || depthMap[neighbor_x * width + neighbor_y] == -INFINITY) {
					continue;
				}
				else
				{
					double N_r = exp(-(pow(sqrt(pow(depthMap[neighbor_x * width + neighbor_y] - depthMap[id_x * width + id_y], 2)), 2)) / pow(sigmaR, 2));

					double N_s = exp(-(pow(sqrt(pow(id_x - neighbor_x, 2) + pow(id_y - neighbor_y, 2)), 2)) / pow(sigmaS, 2));

					double w = N_s * N_r;
					filtered += depthMap[neighbor_x * width + neighbor_y] * w;
					wP = wP + w;
					//atomicAdd(&filtered, depthMap[neighbor_x * width + neighbor_y] * w);
					//atomicAdd(&wP, w);
				}
			}
		}
	}
	if (wP == 0.0) {
		filtered = 0;
	}

	else {
		filtered = filtered / wP;
	}

	filteredImage[id_x * width + id_y] = filtered;
}


__global__
void calculate_Vks_cuda(Eigen::Matrix3f K_i,
	Eigen::Vector3f* dV_k,
	float* Depth_k, int* dMk_0, int* dMk_1,
	int width, int height) {

	int id_x = blockIdx.x * blockDim.x + threadIdx.x;
	int id_y = blockIdx.y * blockDim.y + threadIdx.y;
	Eigen::Vector3f u_dot;

	if (id_y < width && id_x < height) {
		u_dot << id_y, id_x, 1;
		if (Depth_k[id_x * width + id_y] == -INFINITY || Depth_k[id_x * width + id_y] <= 0.0f) {
			dV_k[id_x * width + id_y] = Eigen::Vector3f(-INFINITY, -INFINITY, -INFINITY);
			dMk_0[id_x * width + id_y] = id_x * width + id_y;
		}
		else {
			dV_k[id_x * width + id_y] = Depth_k[id_x * width + id_y] * 255.0f * 255.0f / 5000.0f * K_i * u_dot;
			dMk_1[id_x * width + id_y] = id_x * width + id_y;
		}
	}
}

__global__
void calculate_Nks_cuda(Eigen::Vector3f* dV_k,
	Eigen::Vector3f* dN_k,
	int width, int height)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < height - 1 && j < width - 1) {
		Eigen::Vector3f ans = (dV_k[i * width + j + 1] - dV_k[(i)*width + j]).cross((dV_k[(i + 1) * width + j] - dV_k[(i)*width + j]));
		ans.normalize();
		dN_k[i * width + j] = ans;
	}
	else {
		Eigen::Vector3f ans = (dV_k[i * width + (width - 1) - 1] - dV_k[(i)*width + (width - 1)]).cross((dV_k[(i + 1) * width + (width - 1)] - dV_k[(i)*width + (width - 1)]));
		ans.normalize();
		dN_k[i * width + j] = ans;
	}


}


std::vector<Eigen::Vector3f> Frame::calculate_Vks()
{
	V_k.resize(width * height);

	Eigen::Matrix3f K_i = K_calibration.inverse();

	Eigen::Vector3f* dV_k;

	int* dMk_1;
	int* dMk_0;

	//int* M_k1_new = new int[height * width];
	M_k0.resize(width * height);
	M_k1.resize(width * height);
	float* filtered_img_gpu;

	hipError_t cudaStatus1 = hipMalloc(&dV_k, height * width * sizeof(Eigen::Vector3f));
	if (cudaStatus1 != hipSuccess) {
		std::cout << "Problem in memory allocation: " << hipGetErrorString(cudaStatus1) << std::endl;
	};
	hipError_t cudaStatus4 = hipMalloc(&dMk_0, height * width * sizeof(float));
	if (cudaStatus4 != hipSuccess) {
		std::cout << "Problem in memory allocation: " << hipGetErrorString(cudaStatus4) << std::endl;
	};
	hipError_t cudaStatus3 = hipMalloc(&dMk_1, width * height * sizeof(float));
	if (cudaStatus3 != hipSuccess) {
		std::cout << "Problem in memory allocation: " << hipGetErrorString(cudaStatus3) << std::endl;
	};

	hipError_t cudaStatus10 = hipMalloc(&filtered_img_gpu, height * width * sizeof(float));
	if (cudaStatus10 != hipSuccess) {
		std::cout << "Problem in memory allocation: " << hipGetErrorString(cudaStatus10) << std::endl;
	};
	hipError_t cudaStatus0 = hipMemcpy(filtered_img_gpu, Depth_k, width * height * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus3 != hipSuccess) {
		std::cout << "Problem in Copying1212: " << hipGetErrorString(cudaStatus3) << std::endl;
	};

	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(height / threadsPerBlock.x, width / threadsPerBlock.y);
	calculate_Vks_cuda << <numBlocks, threadsPerBlock >> > (K_i, dV_k, filtered_img_gpu, dMk_0, dMk_1, width, height);
	hipDeviceSynchronize();


	hipError_t cudaStatus2 = hipMemcpy(V_k.data(), dV_k, width * height * sizeof(Eigen::Vector3f), hipMemcpyDeviceToHost);
	if (cudaStatus2 != hipSuccess) {
		std::cout << "Problem in Copying1313: " << hipGetErrorString(cudaStatus2) << std::endl;
	};
	hipError_t cudaStatus5 = hipMemcpy(M_k1.data(), dMk_1, width * height * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus5 != hipSuccess) {
		std::cout << "Problem in Copying1414: " << hipGetErrorString(cudaStatus5) << std::endl;
	};
	hipError_t cudaStatus6 = hipMemcpy(M_k0.data(), dMk_0, width * height * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus6 != hipSuccess) {
		std::cout << "Problem in Copying1415: " << hipGetErrorString(cudaStatus6) << std::endl;
	};
	hipFree(dV_k);
	hipFree(dMk_1);
	hipFree(dMk_0);
	hipFree(filtered_img_gpu);

	return V_k;
}

std::vector<Eigen::Vector3f>  Frame::calculate_Nks()
{
	N_k.resize(width * height);
	Eigen::Vector3f* dN_k;
	Eigen::Vector3f* V_k_array;

	hipError_t cudaStatus1 = hipMalloc(&dN_k, height * width * sizeof(Eigen::Vector3f));
	if (cudaStatus1 != hipSuccess) {
		std::cout << "Problem in memory allocation: " << hipGetErrorString(cudaStatus1) << std::endl;
	};
	hipError_t cudaStatus2 = hipMalloc(&V_k_array, height * width * sizeof(Eigen::Vector3f));
	if (cudaStatus2 != hipSuccess) {
		std::cout << "Problem in memory allocation: " << hipGetErrorString(cudaStatus2) << std::endl;
	};
	hipError_t cudaStatus3 = hipMemcpy(V_k_array, V_k.data(), width * height * sizeof(Eigen::Vector3f), hipMemcpyHostToDevice);
	if (cudaStatus3 != hipSuccess) {
		std::cout << "Problem in Copying3: " << hipGetErrorString(cudaStatus3) << std::endl;
	};

	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(height / threadsPerBlock.x, width / threadsPerBlock.y);
	calculate_Nks_cuda << <numBlocks, threadsPerBlock >> > (V_k_array, dN_k, width, height);
	hipDeviceSynchronize();


	hipError_t cudaStatus4 = hipMemcpy(N_k.data(), dN_k, width * height * sizeof(Eigen::Vector3f), hipMemcpyDeviceToHost);
	if (cudaStatus4 != hipSuccess) {
		std::cout << "Problem in Copying4: " << hipGetErrorString(cudaStatus4) << std::endl;
	};

	hipFree(dN_k);
	hipFree(V_k_array);

	return N_k;
}

float* Frame::bilateralFilter_cu(int diameter, double sigmaS, double sigmaR) {
	float* depthMap = new float[height * width];
	float* filteredImage = new float[height * width];
	float* filteredImage_final = new float[height * width];


	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(height / threadsPerBlock.x, width / threadsPerBlock.y);

	hipError_t cudaStatus1 = hipMalloc(&filteredImage, height * width * sizeof(float));
	if (cudaStatus1 != hipSuccess) {
		std::cout << "Problem in memory allocation: " << hipGetErrorString(cudaStatus1) << std::endl;
	};
	hipError_t cudaStatus4 = hipMalloc(&depthMap, height * width * sizeof(float));
	if (cudaStatus4 != hipSuccess) {
		std::cout << "Problem in memory allocation: " << hipGetErrorString(cudaStatus4) << std::endl;
	};
	hipError_t cudaStatus3 = hipMemcpy(depthMap, Raw_k, width * height * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus3 != hipSuccess) {
		std::cout << "Problem in Copying: " << hipGetErrorString(cudaStatus3) << std::endl;
	};


	apply_bilateral_cuda << <numBlocks, threadsPerBlock >> > (depthMap, filteredImage, diameter, sigmaS, sigmaR, width, height);
	hipDeviceSynchronize();

	hipError_t cudaStatus2 = hipMemcpy(filteredImage_final, filteredImage, width * height * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus2 != hipSuccess) {
		std::cout << "Problem in Copying: " << hipGetErrorString(cudaStatus2) << std::endl;
	};
	hipFree(filteredImage);
	hipFree(depthMap);
	//filteredImage_final = symmetricImageX(filteredImage_final, width, height);

	return filteredImage_final;
}

void Frame::save_off_format(const std::string& where_to_save) {

	std::ofstream OffFile(where_to_save);
	for (int j = 0; j < width * height; j++) {
		int i = M_k1[j];
		if (i == 0) {
			continue;
		}
		if (abs(V_k[i][0]) < MAXTHRESHOLD) {
			OffFile << "v " << V_k[i][0] << " " << V_k[i][1] << " " << V_k[i][2] << std::endl;
			if (!std::isnan(N_k[i][0]) && !std::isnan(N_k[i][1]) && !std::isnan(N_k[i][2])) {
				OffFile << "vn " << N_k[i][0] << " " << N_k[i][1] << " " << N_k[i][2] << std::endl;
			}
			else {
				OffFile << "vn " << 0 << " " << 0 << " " << 0 << std::endl;
			}
		}
	}
	OffFile.close();
}

Frame::Frame(FIBITMAP& dib, Eigen::Matrix4f T_gk, float sub_sampling_rate) :
	dib(FreeImage_ConvertToFloat(&dib)), T_gk(T_gk) {

	width = FreeImage_GetWidth(this->dib);
	height = FreeImage_GetHeight(this->dib);

	Depth_k = new float[width * height]; // have to rescale according to the data 

	Raw_k = (float*)FreeImage_GetBits(this->dib); // have to rescale according to the data 

	K_calibration << 525.0f / sub_sampling_rate, 0.0f, 319.5f / sub_sampling_rate,
		0.0f, 525.0f / sub_sampling_rate, 239.5f / sub_sampling_rate,
		0.0f, 0.0f, 1.0f;
}

Frame::Frame(const char* image_dir, Eigen::Matrix4f T_gk, float sub_sampling_rate) :
	dib(FreeImage_ConvertToFloat(FreeImage_Load(FreeImage_GetFileType(image_dir), image_dir))) {

	// FreeImage_Initialise();

	width = FreeImage_GetWidth(this->dib);
	height = FreeImage_GetHeight(this->dib);

	Depth_k = new float[width * height]; // have to rescale according to the data 

	Raw_k = (float*)FreeImage_GetBits(this->dib); // have to rescale according to the data 

	K_calibration << 525.0f / sub_sampling_rate, 0.0f, 319.5f / sub_sampling_rate,
		0.0f, 525.0f / sub_sampling_rate, 239.5f / sub_sampling_rate,
		0.0f, 0.0f, 1.0f;

	this->T_gk = T_gk;

	// FreeImage_DeInitialise();
}

Frame::~Frame() {
	
	if (dib != nullptr) { 
		FreeImage_Unload(dib);
		dib = nullptr;
	}
	if (Depth_k != nullptr) { 
		delete[] Depth_k;
		Depth_k = nullptr;
	}
	if(Raw_k != nullptr){
//		FreeImage_Unload((FITBITMAP *)Raw_k);
		Raw_k = nullptr;
	}
	if(filtered_dib != nullptr){
		FreeImage_Unload(filtered_dib);
		filtered_dib = nullptr;
	}

}

void Frame::process_image(float sigma_r, float sigma_s, int filter_size, bool apply_bilateral) {
	
	if (apply_bilateral) {
		Depth_k = bilateralFilter_cu(15, 3.0, 0.01);
	}
	else
	{
		Depth_k = Raw_k;
	}
	// cuda
	calculate_Vks();
	calculate_Nks();
	//save_off_format("C:/Users/yigitavci/Desktop/TUM_DERS/Semester_2/3D_Scanning/KinectFusion-Cool-Edition/scene2_cudaa_nofilter.obj");

}

Frame::Frame(std::vector<Eigen::Vector3f> V_gks, std::vector<Eigen::Vector3f> N_gks, Eigen::Matrix4f T_gk, int width, int height):
width(width), height(height), T_gk(T_gk), V_gk(V_gks), N_gk(N_gks){
    K_calibration  <<  525.0f , 0.0f, 319.5f,
                        0.0f, 525.0f, 239.5f,
                        0.0f, 0.0f, 1.0f;
	transformed = true;
    
}

void Frame::save_G_off_format(const std::string & where_to_save)
{
        std::ofstream OffFile(where_to_save);
        this -> apply_G_transform();
        for(unsigned int i = 0; i < width * height; ++i){
            if(abs(V_gk[i][0]) < MAXTHRESHOLD){
                if (V_gk[i][0] != MINF)
                {
                    OffFile << "v " << V_gk[i][0] << " " << V_gk[i][1] << " " << V_gk[i][2] << std::endl; 
                    if(!std::isnan(N_gk[i][0]) && !std::isnan(N_gk[i][1]) && !std::isnan(N_gk[i][2])){
                        OffFile << "vn " << N_gk[i][0] << " " << N_gk[i][1] << " " << N_gk[i][2] << std::endl;
                    }
                    else{
                        OffFile << "vn " << 0 << " " << 0 << " " << 0 << std::endl;
                    } 
                }
            }
        }
        OffFile.close();
    }
