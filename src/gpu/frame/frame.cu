#include "hip/hip_runtime.h"
#include "../../cpu/frame/Frame.h"
#include "../../cpu/frame/Frame_Pyramid.h"
#include <iostream>
#include <fstream>
#include <FreeImage.h>
#include <vector>
#include <eigen3/Eigen/Dense>
#define MINF -std::numeric_limits<float>::infinity()

#define MAXTHRESHOLD 10

__global__
void apply_bilateral_cuda(float* depthMap, float* filteredImage, int diameter, double sigmaS, double sigmaR, int width, int height) {
	double filtered = 0;
	double wP = 0;
	int neighbor_x = 0;
	int neighbor_y = 0;
	int half = diameter / 2;
	int id_x = blockIdx.x * blockDim.x + threadIdx.x;
	int id_y = blockIdx.y * blockDim.y + threadIdx.y;
	for (int i = 0; i < diameter; i++) {
		for (int j = 0; j < diameter; j++) {

			neighbor_x = id_x - (half - i);
			neighbor_y = id_y - (half - j);
			if (neighbor_x > 0 && neighbor_y > 0 && neighbor_x < height && neighbor_y < width)
			{
				if (depthMap[neighbor_x * width + neighbor_y] <= 0.0f || depthMap[neighbor_x * width + neighbor_y] == -INFINITY) {
					continue;
				}
				else
				{
					double N_r = exp(-(pow(sqrt(pow(depthMap[neighbor_x * width + neighbor_y] - depthMap[id_x * width + id_y], 2)), 2)) / pow(sigmaR, 2));

					double N_s = exp(-(pow(sqrt(pow(id_x - neighbor_x, 2) + pow(id_y - neighbor_y, 2)), 2)) / pow(sigmaS, 2));

					double w = N_s * N_r;
					filtered += depthMap[neighbor_x * width + neighbor_y] * w;
					wP = wP + w;
					//atomicAdd(&filtered, depthMap[neighbor_x * width + neighbor_y] * w);
					//atomicAdd(&wP, w);
				}
			}
		}
	}
	if (wP == 0.0) {
		filtered = 0;
	}

	else {
		filtered = filtered / wP;
	}

	filteredImage[id_x * width + id_y] = filtered;
}


__global__
void calculate_Vks_cuda(Eigen::Matrix3f K_i,
	Eigen::Vector3f* dV_k,
	float* Depth_k, int* dMk_0, int* dMk_1,
	int width, int height, bool online) {

	int id_x = threadIdx.x; // pixels per row
    int id_y = blockIdx.x;  // rows
	
	Eigen::Vector3f u_dot;

	 if(id_x < width && id_y < height) {
		int i = id_y * width + id_x;
		u_dot << id_x, id_y, 1;
		if (Depth_k[i] == -INFINITY || Depth_k[i] <= 0.0f) {
			dV_k[i] = Eigen::Vector3f(-INFINITY, -INFINITY, -INFINITY);
			dMk_0[i] = i;
		}
		else {
			// printf("I'm here %f", Depth_k[i]);
			if(online){
				dV_k[i] = Depth_k[i] * K_i * u_dot;
				dMk_1[i] = i;
			}
			else {
				dV_k[i] = Depth_k[i] * 255.0f * 255.0f / 5000.0f *K_i * u_dot;
				dMk_1[i] = i;
			}
			
		}
	}
}

__global__
void calculate_Nks_cuda(Eigen::Vector3f* dV_k,
	Eigen::Vector3f* dN_k,
	int width, int height)
{

	int id_x = threadIdx.x; // pixels per row
    int id_y = blockIdx.x;  // rows
	if(id_x < width - 1 && id_y < height - 1){
		
		int idx_vector = id_y * width + id_x;
		int idx_vector_right = id_y * width + id_x + 1;
		int idx_vector_down = (id_y + 1) * width + id_x;

		Eigen::Vector3f ans = (dV_k[idx_vector_right] - dV_k[idx_vector]).cross((dV_k[idx_vector_down] - dV_k[idx_vector]));
		ans.normalize();
		dN_k[idx_vector] = ans;
	}
	else if(id_x == width - 1 && id_y < height - 1) {

		int idx_vector = id_y * width + id_x;
		int idx_vector_left = id_y * width + id_x - 1;
		int idx_vector_down = (id_y + 1) * width + id_x;

		Eigen::Vector3f ans = (dV_k[idx_vector_left] - dV_k[idx_vector]).cross((dV_k[idx_vector_down] - dV_k[idx_vector]));
		ans.normalize();
		dN_k[idx_vector] = ans;
	}
	else if(id_x < width - 1 && id_y == height - 1) {

		int idx_vector = id_y * width + id_x;
		int idx_vector_right = id_y * width + id_x + 1;
		int idx_vector_up = (id_y - 1) * width + id_x;

		Eigen::Vector3f ans = (dV_k[idx_vector_right] - dV_k[idx_vector]).cross((dV_k[idx_vector_up] - dV_k[idx_vector]));
		ans.normalize();
		dN_k[idx_vector] = ans;
	}
	else if(id_x == width - 1 && id_y == height - 1) {

		int idx_vector = id_y * width + id_x;
		int idx_vector_left = id_y * width + id_x - 1;
		int idx_vector_up = (id_y - 1) * width + id_x;

		Eigen::Vector3f ans = (dV_k[idx_vector_left] - dV_k[idx_vector]).cross((dV_k[idx_vector_up] - dV_k[idx_vector]));
		ans.normalize();
		dN_k[idx_vector] = ans;
	}


}


std::vector<Eigen::Vector3f> Frame::calculate_Vks()
{
	V_k.resize(width * height);

	Eigen::Matrix3f K_i = K_calibration.inverse();

	Eigen::Vector3f* dV_k;

	int* dMk_1;
	int* dMk_0;

	//int* M_k1_new = new int[height * width];
	M_k0.resize(width * height);
	M_k1.resize(width * height);
	float* filtered_img_gpu;

	hipError_t cudaStatus1 = hipMallocManaged(&dV_k, height * width * sizeof(Eigen::Vector3f));
	if (cudaStatus1 != hipSuccess) {
		std::cout << "Problem in memory allocation: " << hipGetErrorString(cudaStatus1) << std::endl;
	};
	hipError_t cudaStatus4 = hipMallocManaged(&dMk_0, height * width * sizeof(float));
	if (cudaStatus4 != hipSuccess) {
		std::cout << "Problem in memory allocation: " << hipGetErrorString(cudaStatus4) << std::endl;
	};
	hipError_t cudaStatus3 = hipMallocManaged(&dMk_1, width * height * sizeof(float));
	if (cudaStatus3 != hipSuccess) {
		std::cout << "Problem in memory allocation: " << hipGetErrorString(cudaStatus3) << std::endl;
	};

	hipError_t cudaStatus10 = hipMallocManaged(&filtered_img_gpu, height * width * sizeof(float));
	if (cudaStatus10 != hipSuccess) {
		std::cout << "Problem in memory allocation: " << hipGetErrorString(cudaStatus10) << std::endl;
	};
	hipError_t cudaStatus0 = hipMemcpy(filtered_img_gpu, Depth_k, width * height * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus3 != hipSuccess) {
		std::cout << "Problem in Copying1212: " << hipGetErrorString(cudaStatus3) << std::endl;
	};

	int block_num = height;
    int thread_num = width;
	
	calculate_Vks_cuda <<< block_num, thread_num >>> (K_i, dV_k, filtered_img_gpu, dMk_0, dMk_1, width, height, true);
	// hipDeviceSynchronize();

	hipError_t cudaStatus2 = hipMemcpy(V_k.data(), dV_k, width * height * sizeof(Eigen::Vector3f), hipMemcpyDeviceToHost);
	if (cudaStatus2 != hipSuccess) {
		std::cout << "Problem in Copying1313: " << hipGetErrorString(cudaStatus2) << std::endl;
	};
	hipError_t cudaStatus5 = hipMemcpy(M_k1.data(), dMk_1, width * height * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus5 != hipSuccess) {
		std::cout << "Problem in Copying1414: " << hipGetErrorString(cudaStatus5) << std::endl;
	};
	hipError_t cudaStatus6 = hipMemcpy(M_k0.data(), dMk_0, width * height * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus6 != hipSuccess) {
		std::cout << "Problem in Copying1415: " << hipGetErrorString(cudaStatus6) << std::endl;
	};
	hipFree(dV_k);
	hipFree(dMk_1);
	hipFree(dMk_0);
	hipFree(filtered_img_gpu);

	return V_k;
}

std::vector<Eigen::Vector3f>  Frame::calculate_Nks()
{
	N_k.resize(width * height);
	Eigen::Vector3f* dN_k;
	Eigen::Vector3f* V_k_array;

	hipError_t cudaStatus1 = hipMallocManaged(&dN_k, height * width * sizeof(Eigen::Vector3f));
	if (cudaStatus1 != hipSuccess) {
		std::cout << "Problem in memory allocation: " << hipGetErrorString(cudaStatus1) << std::endl;
	};
	hipError_t cudaStatus2 = hipMallocManaged(&V_k_array, height * width * sizeof(Eigen::Vector3f));
	if (cudaStatus2 != hipSuccess) {
		std::cout << "Problem in memory allocation: " << hipGetErrorString(cudaStatus2) << std::endl;
	};
	hipError_t cudaStatus3 = hipMemcpy(V_k_array, V_k.data(), width * height * sizeof(Eigen::Vector3f), hipMemcpyHostToDevice);
	if (cudaStatus3 != hipSuccess) {
		std::cout << "Problem in Copying3: " << hipGetErrorString(cudaStatus3) << std::endl;
	};

	int block_num = height;
    int thread_num = width;
	calculate_Nks_cuda <<<block_num, thread_num >>> (V_k_array, dN_k, width, height);
	hipDeviceSynchronize();


	hipError_t cudaStatus4 = hipMemcpy(N_k.data(), dN_k, width * height * sizeof(Eigen::Vector3f), hipMemcpyDeviceToHost);
	if (cudaStatus4 != hipSuccess) {
		std::cout << "Problem in Copying4: " << hipGetErrorString(cudaStatus4) << std::endl;
	};

	hipFree(dN_k);
	hipFree(V_k_array);

	return N_k;
}

float* Frame::bilateralFilter_cu(int diameter, double sigmaS, double sigmaR) {
	float* depthMap = new float[height * width];
	float* filteredImage = new float[height * width];
	float* filteredImage_final = new float[height * width];


	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(height / threadsPerBlock.x, width / threadsPerBlock.y);

	hipError_t cudaStatus1 = hipMallocManaged(&filteredImage, height * width * sizeof(float));
	if (cudaStatus1 != hipSuccess) {
		std::cout << "Problem in memory allocation: " << hipGetErrorString(cudaStatus1) << std::endl;
	};
	hipError_t cudaStatus4 = hipMallocManaged(&depthMap, height * width * sizeof(float));
	if (cudaStatus4 != hipSuccess) {
		std::cout << "Problem in memory allocation: " << hipGetErrorString(cudaStatus4) << std::endl;
	};
	hipError_t cudaStatus3 = hipMemcpy(depthMap, Depth_k, width * height * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus3 != hipSuccess) {
		std::cout << "Problem in Copying: " << hipGetErrorString(cudaStatus3) << std::endl;
	};


	apply_bilateral_cuda <<< numBlocks, threadsPerBlock >>> (depthMap, filteredImage, diameter, sigmaS, sigmaR, width, height);
	hipDeviceSynchronize();

	hipError_t cudaStatus2 = hipMemcpy(filteredImage_final, filteredImage, width * height * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus2 != hipSuccess) {
		std::cout << "Problem in Copying: " << hipGetErrorString(cudaStatus2) << std::endl;
	};
	hipFree(filteredImage);
	hipFree(depthMap);
	//filteredImage_final = symmetricImageX(filteredImage_final, width, height);

	return filteredImage_final;
}

void Frame::save_off_format(const std::string& where_to_save) {

	std::ofstream OffFile(where_to_save);
	for (int j = 0; j < width * height; j++) {
		int i = M_k1[j];
		if (i == 0) {
			continue;
		}
		if (abs(V_k[i][0]) < MAXTHRESHOLD) {
			OffFile << "v " << V_k[i][0] << " " << V_k[i][1] << " " << V_k[i][2] << std::endl;
			if (!std::isnan(N_k[i][0]) && !std::isnan(N_k[i][1]) && !std::isnan(N_k[i][2])) {
				OffFile << "vn " << N_k[i][0] << " " << N_k[i][1] << " " << N_k[i][2] << std::endl;
			}
			else {
				OffFile << "vn " << 0 << " " << 0 << " " << 0 << std::endl;
			}
		}
	}
	OffFile.close();
}

Frame::Frame(FIBITMAP& dib, Eigen::Matrix4f T_gk, float sub_sampling_rate) :
	dib(FreeImage_ConvertToFloat(&dib)), T_gk(T_gk) {

	width = FreeImage_GetWidth(this->dib);
	height = FreeImage_GetHeight(this->dib);

	Depth_k = new float[width * height]; // have to rescale according to the data 

	Raw_k = (float*)FreeImage_GetBits(this->dib); // have to rescale according to the data 

	K_calibration << 525.0f / sub_sampling_rate, 0.0f, 319.5f / sub_sampling_rate,
		0.0f, 525.0f / sub_sampling_rate, 239.5f / sub_sampling_rate,
		0.0f, 0.0f, 1.0f;
}

Frame::Frame(std::vector<float> depthMap, Eigen::Matrix4f T_gk, Eigen::Matrix3f K, int width, int height, float sub_sampling_rate) :
	dib(nullptr), T_gk(T_gk), K_calibration(K), Raw_k(depthMap.data()), width(width), height(height) {
	// live sensor constructor
	K(0,0) /= sub_sampling_rate;
	K(0,2) /= sub_sampling_rate;
	K(1,1) /= sub_sampling_rate;
	K(1,2) /= sub_sampling_rate;
	
	Depth_k = new float[width * height]; // have to rescale according to the data 
	for(int i = 0; i < width*height; i++){
		Depth_k[i] = depthMap[i];
	}
}

Frame::Frame(const char* image_dir, Eigen::Matrix4f T_gk, float sub_sampling_rate) :
	dib(FreeImage_ConvertToFloat(FreeImage_Load(FreeImage_GetFileType(image_dir), image_dir))) {

	// FreeImage_Initialise();

	width = FreeImage_GetWidth(this->dib);
	height = FreeImage_GetHeight(this->dib);

	Depth_k = new float[width * height]; // have to rescale according to the data 

	Raw_k = (float*)FreeImage_GetBits(this->dib); // have to rescale according to the data 

	K_calibration << 525.0f / sub_sampling_rate, 0.0f, 319.5f / sub_sampling_rate,
		0.0f, 525.0f / sub_sampling_rate, 239.5f / sub_sampling_rate,
		0.0f, 0.0f, 1.0f;

	this->T_gk = T_gk;

	// FreeImage_DeInitialise();
}

Frame::~Frame() {
	
	if (dib != nullptr) { 
		FreeImage_Unload(dib);
		dib = nullptr;
	}
	if (Depth_k != nullptr) { 
		delete[] Depth_k;
		Depth_k = nullptr;
	}
	if(Raw_k != nullptr){
//		FreeImage_Unload((FITBITMAP *)Raw_k);
		Raw_k = nullptr;
	}
	if(filtered_dib != nullptr){
		FreeImage_Unload(filtered_dib);
		filtered_dib = nullptr;
	}

}

void Frame::process_image(float sigma_r, float sigma_s, int filter_size, bool apply_bilateral) {
	
	if (apply_bilateral) {
		Depth_k = bilateralFilter_cu(3, 3.0, 0.01);
	}
	else
	{
		Depth_k = Raw_k;
	}
	// cuda
	calculate_Vks();
	calculate_Nks();
	//save_off_format("C:/Users/yigitavci/Desktop/TUM_DERS/Semester_2/3D_Scanning/KinectFusion-Cool-Edition/scene2_cudaa_nofilter.obj");

}

Frame::Frame(std::vector<Eigen::Vector3f> V_gks, std::vector<Eigen::Vector3f> N_gks, Eigen::Matrix4f T_gk, int width, int height):
width(width), height(height), T_gk(T_gk), V_gk(V_gks), N_gk(N_gks){
    K_calibration  <<  525.0f , 0.0f, 319.5f,
                        0.0f, 525.0f, 239.5f,
                        0.0f, 0.0f, 1.0f;
	transformed = true;
    
}

void Frame::save_G_off_format(const std::string & where_to_save)
{
        std::ofstream OffFile(where_to_save);
        this -> apply_G_transform();
        for(unsigned int i = 0; i < width * height; ++i){
            if(abs(V_gk[i][0]) < MAXTHRESHOLD){
                if (V_gk[i][0] != MINF)
                {
                    OffFile << "v " << V_gk[i][0] << " " << V_gk[i][1] << " " << V_gk[i][2] << std::endl; 
                    if(!std::isnan(N_gk[i][0]) && !std::isnan(N_gk[i][1]) && !std::isnan(N_gk[i][2])){
                        OffFile << "vn " << N_gk[i][0] << " " << N_gk[i][1] << " " << N_gk[i][2] << std::endl;
                    }
                    else{
                        OffFile << "vn " << 0 << " " << 0 << " " << 0 << std::endl;
                    } 
                }
            }
        }
        OffFile.close();
    }
